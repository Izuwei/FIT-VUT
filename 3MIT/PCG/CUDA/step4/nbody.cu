#include "hip/hip_runtime.h"
/**
 * @File nbody.cu
 *
 * Implementation of the N-Body problem
 *
 * Paralelní programování na GPU (PCG 2021)
 * Projekt c. 1 (cuda)
 * Login: xsadil07
 */

#include <cmath>
#include <cfloat>
#include "nbody.h"

/**
 * CUDA kernel to calculate velocity
 * @param p_in    - particles (input)
 * @param p_out   - particles (output)
 * @param N       - Number of particles
 * @param dt      - Size of the time step
 */
__global__ void calculate_velocity(t_particles p_in, t_particles p_out, int N, float dt)
{
  extern __shared__ float shared_memory[];

  // Setup pointers to shared memory
  t_particles cache {
    .weights = shared_memory,
    .positions_x = &shared_memory[blockDim.x],
    .positions_y = &shared_memory[blockDim.x * 2],
    .positions_z = &shared_memory[blockDim.x * 3],
    .velocities_x = &shared_memory[blockDim.x * 4],
    .velocities_y = &shared_memory[blockDim.x * 5],
    .velocities_z = &shared_memory[blockDim.x * 6],
  };

  unsigned int g_id = threadIdx.x + blockIdx.x * blockDim.x;

  // Setup registers to avoid repeated access into global memory
  float F, r, dx, dy, dz;

  float vx = 0;
  float vy = 0;
  float vz = 0;

  float p1_w = p_in.weights[g_id];
  float p1_x = p_in.positions_x[g_id];
  float p1_y = p_in.positions_y[g_id];
  float p1_z = p_in.positions_z[g_id];
  float p1_vx = p_in.velocities_x[g_id];
  float p1_vy = p_in.velocities_y[g_id];
  float p1_vz = p_in.velocities_z[g_id];

  unsigned int offset;
  unsigned int tile_size = blockDim.x;
  unsigned int tile_cnt = ceil(float(N) / tile_size);

  // Loop over tiles
  for (unsigned int tile_id = 0; tile_id < tile_cnt; tile_id++) {
    offset = threadIdx.x + tile_id * blockDim.x;

    // Load data into shared memory
    cache.weights[threadIdx.x] = (offset < N) ? p_in.weights[offset] : 0.0f;
    cache.positions_x[threadIdx.x] = (offset < N) ? p_in.positions_x[offset] : 0.0f;
    cache.positions_y[threadIdx.x] = (offset < N) ? p_in.positions_y[offset] : 0.0f;
    cache.positions_z[threadIdx.x] = (offset < N) ? p_in.positions_z[offset] : 0.0f;
    cache.velocities_x[threadIdx.x] = (offset < N) ? p_in.velocities_x[offset] : 0.0f;
    cache.velocities_y[threadIdx.x] = (offset < N) ? p_in.velocities_y[offset] : 0.0f;
    cache.velocities_z[threadIdx.x] = (offset < N) ? p_in.velocities_z[offset] : 0.0f;

    __syncthreads();

    // Loop over particles in tile
    for (unsigned p_id = 0; p_id < tile_size; p_id++) {
      dx = p1_x - cache.positions_x[p_id];
      dy = p1_y - cache.positions_y[p_id];
      dz = p1_z - cache.positions_z[p_id];

      r = sqrt(dx*dx + dy*dy + dz*dz);

      // Gravitation velocity
      F = -G * p1_w * cache.weights[p_id] / (r * r + FLT_MIN);

      vx += (r > COLLISION_DISTANCE) ? ((F * dx / (r + FLT_MIN)) * dt / p1_w) : 0.0f;
      vy += (r > COLLISION_DISTANCE) ? ((F * dy / (r + FLT_MIN)) * dt / p1_w) : 0.0f;
      vz += (r > COLLISION_DISTANCE) ? ((F * dz / (r + FLT_MIN)) * dt / p1_w) : 0.0f;

      // Collision velocity
      vx += (r > 0.0f && r < COLLISION_DISTANCE) ? (((p1_w * p1_vx - cache.weights[p_id] * p1_vx + 2 * cache.weights[p_id] * cache.velocities_x[p_id]) /
            (p1_w + cache.weights[p_id])) - p1_vx) : 0.0f;
      vy += (r > 0.0f && r < COLLISION_DISTANCE) ? (((p1_w * p1_vy - cache.weights[p_id] * p1_vy + 2 * cache.weights[p_id] * cache.velocities_y[p_id]) /
            (p1_w + cache.weights[p_id])) - p1_vy) : 0.0f;
      vz += (r > 0.0f && r < COLLISION_DISTANCE) ? (((p1_w * p1_vz - cache.weights[p_id] * p1_vz + 2 * cache.weights[p_id] * cache.velocities_z[p_id]) /
            (p1_w + cache.weights[p_id])) - p1_vz) : 0.0f;
    }
    __syncthreads();
  }

  // Update particles
  if (g_id < N) {
    p_out.velocities_x[g_id] = p1_vx + vx;
    p_out.velocities_y[g_id] = p1_vy + vy;
    p_out.velocities_z[g_id] = p1_vz + vz;

    p_out.positions_x[g_id] = p1_x + p_out.velocities_x[g_id] * dt;
    p_out.positions_y[g_id] = p1_y + p_out.velocities_y[g_id] * dt;
    p_out.positions_z[g_id] = p1_z + p_out.velocities_z[g_id] * dt;
  }
}// end of calculate_velocity
//----------------------------------------------------------------------------------------------------------------------

/**
 * CUDA kernel to update particles
 * @param p       - particles
 * @param comX    - pointer to a center of mass position in X
 * @param comY    - pointer to a center of mass position in Y
 * @param comZ    - pointer to a center of mass position in Z
 * @param comW    - pointer to a center of mass weight
 * @param lock    - pointer to a user-implemented lock
 * @param N       - Number of particles
 */
__global__ void centerOfMass(t_particles p, float* comX, float* comY, float* comZ, float* comW, int* lock, const int N)
{
  extern __shared__ float shared_memory[];

  // Setup pointers to shared memory
  t_particles cache {
    .weights = shared_memory,
    .positions_x = &shared_memory[blockDim.x],
    .positions_y = &shared_memory[blockDim.x * 2],
    .positions_z = &shared_memory[blockDim.x * 3],
  };

  unsigned int g_id = (blockIdx.x * blockDim.x + threadIdx.x);

  // Initialize values in shared memory
  cache.positions_x[threadIdx.x] = p.positions_x[g_id];
  cache.positions_y[threadIdx.x] = p.positions_y[g_id];
  cache.positions_z[threadIdx.x] = p.positions_z[g_id];
  cache.weights[threadIdx.x] = p.weights[g_id];

  float dx, dy, dz, dw;

  __syncthreads();

  // Reduction within a block
  for (unsigned int i = 0; i < blockDim.x; i++) {
    if (threadIdx.x < i) {
      dx = cache.positions_x[i] - cache.positions_x[threadIdx.x];
      dy = cache.positions_y[i] - cache.positions_y[threadIdx.x];
      dz = cache.positions_z[i] - cache.positions_z[threadIdx.x];
      dw = ((cache.weights[i] + cache.weights[threadIdx.x]) > 0.0f) 
           ? (cache.weights[i] / (cache.weights[i] + cache.weights[threadIdx.x])) : 0.0f;

      cache.positions_x[threadIdx.x] += dx * dw;
      cache.positions_y[threadIdx.x] += dy * dw;
      cache.positions_z[threadIdx.x] += dz * dw;
      cache.weights[threadIdx.x] += cache.weights[i];
    }
    __syncthreads();
  }

  // Global reduction
  if (threadIdx.x == 0 && g_id < N) {
    // Lock mutex before entering the critical section
    while (atomicCAS(lock, 0, 1) != 0);

    dx = cache.positions_x[0] - *comX;
    dy = cache.positions_y[0] - *comY;
    dz = cache.positions_z[0] - *comZ;
    dw = ((cache.weights[0] + *comW) > 0.0f)
         ? (cache.weights[0] / (cache.weights[0] + *comW)) : 0.0f;

    *comX += dx * dw;
    *comY += dy * dw;
    *comZ += dz * dw;
    *comW += cache.weights[0];

    // Unlock mutex
    atomicExch(lock, 0);
  }
}// end of centerOfMass
//----------------------------------------------------------------------------------------------------------------------

/**
 * CPU implementation of the Center of Mass calculation
 * @param particles - All particles in the system
 * @param N         - Number of particles
 */
__host__ float4 centerOfMassCPU(MemDesc& memDesc)
{
  float4 com = {0, 0, 0, 0};

  for(int i = 0; i < memDesc.getDataSize(); i++)
  {
    // Calculate the vector on the line connecting current body and most recent position of center-of-mass
    const float dx = memDesc.getPosX(i) - com.x;
    const float dy = memDesc.getPosY(i) - com.y;
    const float dz = memDesc.getPosZ(i) - com.z;

    // Calculate weight ratio only if at least one particle isn't massless
    const float dw = ((memDesc.getWeight(i) + com.w) > 0.0f)
                          ? ( memDesc.getWeight(i) / (memDesc.getWeight(i) + com.w)) : 0.0f;

    // Update position and weight of the center-of-mass according to the weight ration and vector
    com.x += dx * dw;
    com.y += dy * dw;
    com.z += dz * dw;
    com.w += memDesc.getWeight(i);
  }
  return com;
}// enf of centerOfMassCPU
//----------------------------------------------------------------------------------------------------------------------
