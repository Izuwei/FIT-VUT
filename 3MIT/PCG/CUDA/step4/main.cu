#include "hip/hip_runtime.h"
/**
 * @File main.cu
 *
 * The main file of the project
 *
 * Paralelní programování na GPU (PCG 2021)
 * Projekt c. 1 (cuda)
 * Login: xsadil07
 */

#include <sys/time.h>
#include <cstdio>
#include <cmath>

#include "nbody.h"
#include "h5Helper.h"

/**
 * Main rotine
 * @param argc
 * @param argv
 * @return
 */
int main(int argc, char **argv)
{
  // Time measurement
  struct timeval t1, t2;

  if (argc != 10)
  {
    printf("Usage: nbody <N> <dt> <steps> <threads/block> <write intesity> <reduction threads> <reduction threads/block> <input> <output>\n");
    exit(1);
  }

  // Number of particles
  const int N           = std::stoi(argv[1]);
  // Length of time step
  const float dt        = std::stof(argv[2]);
  // Number of steps
  const int steps       = std::stoi(argv[3]);
  // Number of thread blocks
  const int thr_blc     = std::stoi(argv[4]);
  // Write frequency
  int writeFreq         = std::stoi(argv[5]);
  // number of reduction threads
  const int red_thr     = std::stoi(argv[6]);
  // Number of reduction threads/blocks
  const int red_thr_blc = std::stoi(argv[7]);

  // Size of the simulation CUDA gird - number of blocks
  const size_t simulationGrid = (N + thr_blc - 1) / thr_blc;
  // Size of the reduction CUDA grid - number of blocks
  const size_t reductionGrid  = (red_thr + red_thr_blc - 1) / red_thr_blc;

  // Log benchmark setup
  printf("N: %d\n", N);
  printf("dt: %f\n", dt);
  printf("steps: %d\n", steps);
  printf("threads/block: %d\n", thr_blc);
  printf("blocks/grid: %lu\n", simulationGrid);
  printf("reduction threads/block: %d\n", red_thr_blc);
  printf("reduction blocks/grid: %lu\n", reductionGrid);

  const size_t recordsNum = (writeFreq > 0) ? (steps + writeFreq - 1) / writeFreq : 0;
  writeFreq = (writeFreq > 0) ?  writeFreq : 0;


  t_particles particles_cpu;

  //////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
  //                            FILL IN: CPU side memory allocation (step 0)                                          //
  //////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
  float *host_particles = (float *)malloc(N * T_PAR_MEM_CNT * sizeof(float));

  particles_cpu.weights = host_particles;
  particles_cpu.positions_x = &host_particles[N];
  particles_cpu.positions_y = &host_particles[N * 2];
  particles_cpu.positions_z = &host_particles[N * 3];
  particles_cpu.velocities_x = &host_particles[N * 4];
  particles_cpu.velocities_y = &host_particles[N * 5];
  particles_cpu.velocities_z = &host_particles[N * 6];

  //////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
  //                              FILL IN: memory layout descriptor (step 0)                                          //
  //////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
  /*
   * Caution! Create only after CPU side allocation
   * parameters:
   *                      Stride of two               Offset of the first
   *  Data pointer        consecutive elements        element in floats,
   *                      in floats, not bytes        not bytes
  */
  MemDesc md(
        particles_cpu.positions_x,    1,  0,          // Postition in X
        particles_cpu.positions_y,    1,  0,          // Postition in Y
        particles_cpu.positions_z,    1,  0,          // Postition in Z
        particles_cpu.velocities_x,   1,  0,          // Velocity in X
        particles_cpu.velocities_y,   1,  0,          // Velocity in Y
        particles_cpu.velocities_z,   1,  0,          // Velocity in Z
        particles_cpu.weights,        1,  0,          // Weight
        N,                                            // Number of particles
        recordsNum);                                  // Number of records in output file

  // Initialisation of helper class and loading of input data
  H5Helper h5Helper(argv[8], argv[9], md);

  try
  {
    h5Helper.init();
    h5Helper.readParticleData();
  }
  catch (const std::exception& e)
  {
    std::cerr<<e.what()<<std::endl;
    return -1;
  }


  //t_particles particles_gpu;

  //////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
  //                                  FILL IN: GPU side memory allocation (step 0)                                    //
  //////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
  std::vector<t_particles> particles_gpu(2);

  float *device_particles;
  for (unsigned int i = 0; i < particles_gpu.size(); i++) {
    hipMalloc<float>(&device_particles, N * T_PAR_MEM_CNT * sizeof(float));

    particles_gpu[i].weights = device_particles;
    particles_gpu[i].positions_x = &device_particles[N];
    particles_gpu[i].positions_y = &device_particles[N * 2];
    particles_gpu[i].positions_z = &device_particles[N * 3];
    particles_gpu[i].velocities_x = &device_particles[N * 4];
    particles_gpu[i].velocities_y = &device_particles[N * 5];
    particles_gpu[i].velocities_z = &device_particles[N * 6];
  }

  //////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
  //                                       FILL IN: memory transfers (step 0)                                         //
  //////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
  for (unsigned int i = 0; i < particles_gpu.size(); i++) {
    hipMemcpy(particles_gpu[i].weights, host_particles, N * T_PAR_MEM_CNT * sizeof(float), hipMemcpyHostToDevice);
  }
  
  gettimeofday(&t1, 0);

  // calculate_velocity params
  dim3 dimGrid(simulationGrid);
  dim3 dimBlock(thr_blc);
  unsigned int memory_size = T_PAR_MEM_CNT * sizeof(float) * dimBlock.x;

  // centerOfMass params
  float4 *host_com = (float4 *)malloc(sizeof(float4));

  float4 *device_com;
  hipMalloc<float4>(&device_com, sizeof(float4));

  int *device_lock;
  hipMalloc<int>(&device_lock, sizeof(int));
  hipMemset(device_lock, 0, sizeof(int));

  dim3 redGridDim(reductionGrid);
  dim3 redBlockDim(red_thr_blc);
  unsigned int red_memory_size = red_thr_blc * 4 * sizeof(float);

  // Create streams
  hipStream_t particle_stream, com_stream;
  hipStreamCreate(&particle_stream);
  hipStreamCreate(&com_stream);

  // Create events
  hipEvent_t com_finished, particles_prev, particles_finished;
  hipEventCreate(&com_finished);
  hipEventCreate(&particles_prev);
  hipEventCreate(&particles_finished);

  unsigned int record_cnt = 0;

  for(int s = 0; s < steps; s++)
  {
    ////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
    //                                       FILL IN: kernels invocation (step 0)                                     //
    ////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
    calculate_velocity<<<dimGrid, dimBlock, memory_size, particle_stream>>>(particles_gpu[s & 1], particles_gpu[(s+1) & 1], N, dt);
    hipEventRecord(particles_finished, particle_stream);

    hipMemsetAsync(device_com, 0, sizeof(float4), com_stream);
    centerOfMass<<<redGridDim, redBlockDim, red_memory_size, com_stream>>>(particles_gpu[s & 1], &device_com->x, &device_com->y,
                                                                           &device_com->z, &device_com->w, device_lock, N);
    hipEventRecord(com_finished, com_stream);

    ////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
    //                                          FILL IN: synchronization  (step 4)                                    //
    ////////////////////////////////////////////////////////////////////////////////////////////////////////////////////

    if (writeFreq > 0 && (s % writeFreq == 0))
    {
      //////////////////////////////////////////////////////////////////////////////////////////////////////////////////
      //                          FILL IN: synchronization and file access logic (step 4)                             //
      //////////////////////////////////////////////////////////////////////////////////////////////////////////////////
      hipStreamWaitEvent(particle_stream, particles_prev, 0);
      hipMemcpyAsync(host_particles, particles_gpu[s & 1].weights, N * T_PAR_MEM_CNT * sizeof(float), hipMemcpyDeviceToHost, particle_stream);
      h5Helper.writeParticleData(record_cnt);

      hipStreamWaitEvent(com_stream, com_finished, 0);
      hipMemcpyAsync(host_com, device_com, sizeof(float4), hipMemcpyDeviceToHost, com_stream);
      h5Helper.writeCom(host_com->x, host_com->y, host_com->z, host_com->w, record_cnt);

      record_cnt++;
    }

    hipStreamWaitEvent(particle_stream, particles_finished, 0);
    hipEventRecord(particles_prev, particle_stream);

    hipEventSynchronize(com_finished);
    hipEventSynchronize(particles_finished);
  }

  hipStreamDestroy(particle_stream);
  hipStreamDestroy(com_stream);

  //////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
  //              FILL IN: invocation of center-of-mass kernel (step 3.1, step 3.2, step 4)                           //
  //////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
  hipMemset(device_com, 0, sizeof(float4));
  centerOfMass<<<redGridDim, redBlockDim, red_memory_size>>>(particles_gpu[steps & 1], &device_com->x, &device_com->y,
                                                               &device_com->z, &device_com->w, device_lock, N);

  hipDeviceSynchronize();

  gettimeofday(&t2, 0);

  // Approximate simulation wall time
  double t = (1000000.0 * (t2.tv_sec - t1.tv_sec) + t2.tv_usec - t1.tv_usec) / 1000000.0;
  printf("Time: %f s\n", t);

  hipMemcpy(host_com, device_com, sizeof(float4), hipMemcpyDeviceToHost);

  //////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
  //                             FILL IN: memory transfers for particle data (step 0)                                 //
  //////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
  float4 comOnGPU = {
    host_com->x, 
    host_com->y, 
    host_com->z, 
    host_com->w
  };

  hipMemcpy(host_particles, particles_gpu[steps & 1].weights, N * T_PAR_MEM_CNT * sizeof(float), hipMemcpyDeviceToHost);

  //////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
  //                        FILL IN: memory transfers for center-of-mass (step 3.1, step 3.2)                         //
  //////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
  float4 comOnCPU = centerOfMassCPU(md);

  std::cout << "Center of mass on CPU:" << std::endl
            << comOnCPU.x <<", "
            << comOnCPU.y <<", "
            << comOnCPU.z <<", "
            << comOnCPU.w
            << std::endl;

  std::cout << "Center of mass on GPU:" << std::endl
            << comOnGPU.x<<", "
            << comOnGPU.y<<", "
            << comOnGPU.z<<", "
            << comOnGPU.w
            << std::endl;

  // Writing final values to the file
  h5Helper.writeComFinal(comOnGPU.x, comOnGPU.y, comOnGPU.z, comOnGPU.w);
  h5Helper.writeParticleDataFinal();

  return 0;
}// end of main
//----------------------------------------------------------------------------------------------------------------------
