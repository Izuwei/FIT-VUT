#include "hip/hip_runtime.h"
/**
 * @File nbody.cu
 *
 * Implementation of the N-Body problem
 *
 * Paralelní programování na GPU (PCG 2021)
 * Projekt c. 1 (cuda)
 * Login: xsadil07
 */

#include <cmath>
#include <cfloat>
#include "nbody.h"

/**
 * CUDA kernel to calculate velocity
 * @param p_in    - particles (input)
 * @param p_out   - particles (output)
 * @param N       - Number of particles
 * @param dt      - Size of the time step
 */
__global__ void calculate_velocity(t_particles p_in, t_particles p_out, int N, float dt)
{
  unsigned int g_id = threadIdx.x + blockIdx.x * blockDim.x;

  if (g_id >= N) {
    return;
  }

  // Setup registers to avoid repeated access into global memory
  float p2_w, F, r, dx, dy, dz;

  float vx = 0;
  float vy = 0;
  float vz = 0;

  float p1_w = p_in.weights[g_id];
  float p1_x = p_in.positions_x[g_id];
  float p1_y = p_in.positions_y[g_id];
  float p1_z = p_in.positions_z[g_id];
  float p1_vx = p_in.velocities_x[g_id];
  float p1_vy = p_in.velocities_y[g_id];
  float p1_vz = p_in.velocities_z[g_id];

  // Loop over particles
  for (unsigned int p_id = 0; p_id < N; p_id++) {
    p2_w = p_in.weights[p_id];

    dx = p1_x - p_in.positions_x[p_id];
    dy = p1_y - p_in.positions_y[p_id];
    dz = p1_z - p_in.positions_z[p_id];

    r = sqrt(dx*dx + dy*dy + dz*dz);

    // Gravitation velocity
    F = -G * p1_w * p2_w / (r * r + FLT_MIN);

    vx += (r > COLLISION_DISTANCE) ? ((F * dx / (r + FLT_MIN)) * dt / p1_w) : 0.0f;
    vy += (r > COLLISION_DISTANCE) ? ((F * dy / (r + FLT_MIN)) * dt / p1_w) : 0.0f;
    vz += (r > COLLISION_DISTANCE) ? ((F * dz / (r + FLT_MIN)) * dt / p1_w) : 0.0f;

    // Collision velocity
    vx += (r > 0.0f && r < COLLISION_DISTANCE) ? (((p1_w * p1_vx - p2_w * p1_vx + 2 * p2_w * p_in.velocities_x[p_id]) /
          (p1_w + p2_w)) - p1_vx) : 0.0f;
    vy += (r > 0.0f && r < COLLISION_DISTANCE) ? (((p1_w * p1_vy - p2_w * p1_vy + 2 * p2_w * p_in.velocities_y[p_id]) /
          (p1_w + p2_w)) - p1_vy) : 0.0f;
    vz += (r > 0.0f && r < COLLISION_DISTANCE) ? (((p1_w * p1_vz - p2_w * p1_vz + 2 * p2_w * p_in.velocities_z[p_id]) /
          (p1_w + p2_w)) - p1_vz) : 0.0f;
  }

  // Update particles
  p_out.velocities_x[g_id] = p1_vx + vx;
  p_out.velocities_y[g_id] = p1_vy + vy;
  p_out.velocities_z[g_id] = p1_vz + vz;

  p_out.positions_x[g_id] = p1_x + p_out.velocities_x[g_id] * dt;
  p_out.positions_y[g_id] = p1_y + p_out.velocities_y[g_id] * dt;
  p_out.positions_z[g_id] = p1_z + p_out.velocities_z[g_id] * dt;
}// end of calculate_velocity
//----------------------------------------------------------------------------------------------------------------------

/**
 * CUDA kernel to update particles
 * @param p       - particles
 * @param comX    - pointer to a center of mass position in X
 * @param comY    - pointer to a center of mass position in Y
 * @param comZ    - pointer to a center of mass position in Z
 * @param comW    - pointer to a center of mass weight
 * @param lock    - pointer to a user-implemented lock
 * @param N       - Number of particles
 */
__global__ void centerOfMass(t_particles p, float* comX, float* comY, float* comZ, float* comW, int* lock, const int N)
{

}// end of centerOfMass
//----------------------------------------------------------------------------------------------------------------------

/**
 * CPU implementation of the Center of Mass calculation
 * @param particles - All particles in the system
 * @param N         - Number of particles
 */
__host__ float4 centerOfMassCPU(MemDesc& memDesc)
{
  float4 com = {0 ,0, 0, 0};

  for(int i = 0; i < memDesc.getDataSize(); i++)
  {
    // Calculate the vector on the line connecting current body and most recent position of center-of-mass
    const float dx = memDesc.getPosX(i) - com.x;
    const float dy = memDesc.getPosY(i) - com.y;
    const float dz = memDesc.getPosZ(i) - com.z;

    // Calculate weight ratio only if at least one particle isn't massless
    const float dw = ((memDesc.getWeight(i) + com.w) > 0.0f)
                          ? ( memDesc.getWeight(i) / (memDesc.getWeight(i) + com.w)) : 0.0f;

    // Update position and weight of the center-of-mass according to the weight ration and vector
    com.x += dx * dw;
    com.y += dy * dw;
    com.z += dz * dw;
    com.w += memDesc.getWeight(i);
  }
  return com;
}// enf of centerOfMassCPU
//----------------------------------------------------------------------------------------------------------------------
