#include "hip/hip_runtime.h"
/**
 * @File main.cu
 *
 * The main file of the project
 *
 * Paralelní programování na GPU (PCG 2021)
 * Projekt c. 1 (cuda)
 * Login: xsadil07
 */

#include <sys/time.h>
#include <cstdio>
#include <cmath>

#include "nbody.h"
#include "h5Helper.h"

/**
 * Main rotine
 * @param argc
 * @param argv
 * @return
 */
int main(int argc, char **argv)
{
  // Time measurement
  struct timeval t1, t2;

  if (argc != 10)
  {
    printf("Usage: nbody <N> <dt> <steps> <threads/block> <write intesity> <reduction threads> <reduction threads/block> <input> <output>\n");
    exit(1);
  }

  // Number of particles
  const int N           = std::stoi(argv[1]);
  // Length of time step
  const float dt        = std::stof(argv[2]);
  // Number of steps
  const int steps       = std::stoi(argv[3]);
  // Number of thread blocks
  const int thr_blc     = std::stoi(argv[4]);
  // Write frequency
  int writeFreq         = std::stoi(argv[5]);
  // number of reduction threads
  const int red_thr     = std::stoi(argv[6]);
  // Number of reduction threads/blocks
  const int red_thr_blc = std::stoi(argv[7]);

  // Size of the simulation CUDA gird - number of blocks
  const size_t simulationGrid = (N + thr_blc - 1) / thr_blc;
  // Size of the reduction CUDA grid - number of blocks
  const size_t reductionGrid  = (red_thr + red_thr_blc - 1) / red_thr_blc;

  // Log benchmark setup
  printf("N: %d\n", N);
  printf("dt: %f\n", dt);
  printf("steps: %d\n", steps);
  printf("threads/block: %d\n", thr_blc);
  printf("blocks/grid: %lu\n", simulationGrid);
  printf("reduction threads/block: %d\n", red_thr_blc);
  printf("reduction blocks/grid: %lu\n", reductionGrid);

  const size_t recordsNum = (writeFreq > 0) ? (steps + writeFreq - 1) / writeFreq : 0;
  writeFreq = (writeFreq > 0) ?  writeFreq : 0;


  t_particles particles_cpu;

  //////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
  //                            FILL IN: CPU side memory allocation (step 0)                                          //
  //////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
  float *host_particles = (float *)malloc(N * T_PAR_MEM_CNT * sizeof(float));

  particles_cpu.weights = host_particles;
  particles_cpu.positions_x = &host_particles[N];
  particles_cpu.positions_y = &host_particles[N * 2];
  particles_cpu.positions_z = &host_particles[N * 3];
  particles_cpu.velocities_x = &host_particles[N * 4];
  particles_cpu.velocities_y = &host_particles[N * 5];
  particles_cpu.velocities_z = &host_particles[N * 6];

  //////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
  //                              FILL IN: memory layout descriptor (step 0)                                          //
  //////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
  /*
   * Caution! Create only after CPU side allocation
   * parameters:
   *                      Stride of two               Offset of the first
   *  Data pointer        consecutive elements        element in floats,
   *                      in floats, not bytes        not bytes
  */
  MemDesc md(
        particles_cpu.positions_x,    1,  0,          // Postition in X
        particles_cpu.positions_y,    1,  0,          // Postition in Y
        particles_cpu.positions_z,    1,  0,          // Postition in Z
        particles_cpu.velocities_x,   1,  0,          // Velocity in X
        particles_cpu.velocities_y,   1,  0,          // Velocity in Y
        particles_cpu.velocities_z,   1,  0,          // Velocity in Z
        particles_cpu.weights,        1,  0,          // Weight
        N,                                            // Number of particles
        recordsNum);                                  // Number of records in output file

  // Initialisation of helper class and loading of input data
  H5Helper h5Helper(argv[8], argv[9], md);

  try
  {
    h5Helper.init();
    h5Helper.readParticleData();
  }
  catch (const std::exception& e)
  {
    std::cerr<<e.what()<<std::endl;
    return -1;
  }


  //t_particles particles_gpu;

  //////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
  //                                  FILL IN: GPU side memory allocation (step 0)                                    //
  //////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
  std::vector<t_particles> particles_gpu(2);

  float *device_particles;
  for (unsigned int i = 0; i < particles_gpu.size(); i++) {
    hipMalloc<float>(&device_particles, N * T_PAR_MEM_CNT * sizeof(float));

    particles_gpu[i].weights = device_particles;
    particles_gpu[i].positions_x = &device_particles[N];
    particles_gpu[i].positions_y = &device_particles[N * 2];
    particles_gpu[i].positions_z = &device_particles[N * 3];
    particles_gpu[i].velocities_x = &device_particles[N * 4];
    particles_gpu[i].velocities_y = &device_particles[N * 5];
    particles_gpu[i].velocities_z = &device_particles[N * 6];
  }

  //////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
  //                                       FILL IN: memory transfers (step 0)                                         //
  //////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
  for (unsigned int i = 0; i < particles_gpu.size(); i++) {
    hipMemcpy(particles_gpu[i].weights, host_particles, N * T_PAR_MEM_CNT * sizeof(float), hipMemcpyHostToDevice);
  }
  
  gettimeofday(&t1, 0);

  dim3 dimGrid(simulationGrid);
  dim3 dimBlock(thr_blc);

  for(int s = 0; s < steps; s++)
  {
    ////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
    //                                       FILL IN: kernels invocation (step 0)                                     //
    ////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
    calculate_velocity<<<dimGrid, dimBlock>>>(particles_gpu[s & 1], particles_gpu[(s+1) & 1], N, dt);

    ////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
    //                                          FILL IN: synchronization  (step 4)                                    //
    ////////////////////////////////////////////////////////////////////////////////////////////////////////////////////

    if (writeFreq > 0 && (s % writeFreq == 0))
    {
      //////////////////////////////////////////////////////////////////////////////////////////////////////////////////
      //                          FILL IN: synchronization and file access logic (step 4)                             //
      //////////////////////////////////////////////////////////////////////////////////////////////////////////////////
    }
  }


  //////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
  //              FILL IN: invocation of center-of-mass kernel (step 3.1, step 3.2, step 4)                           //
  //////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
  hipDeviceSynchronize();

  gettimeofday(&t2, 0);

  // Approximate simulation wall time
  double t = (1000000.0 * (t2.tv_sec - t1.tv_sec) + t2.tv_usec - t1.tv_usec) / 1000000.0;
  printf("Time: %f s\n", t);


  //////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
  //                             FILL IN: memory transfers for particle data (step 0)                                 //
  //////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
  float4 comOnGPU = {};
  hipMemcpy(host_particles, particles_gpu[steps & 1].weights, N * T_PAR_MEM_CNT * sizeof(float), hipMemcpyDeviceToHost);

  //////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
  //                        FILL IN: memory transfers for center-of-mass (step 3.1, step 3.2)                         //
  //////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
  float4 comOnCPU = centerOfMassCPU(md);

  std::cout << "Center of mass on CPU:" << std::endl
            << comOnCPU.x <<", "
            << comOnCPU.y <<", "
            << comOnCPU.z <<", "
            << comOnCPU.w
            << std::endl;

  std::cout << "Center of mass on GPU:" << std::endl
            << comOnGPU.x<<", "
            << comOnGPU.y<<", "
            << comOnGPU.z<<", "
            << comOnGPU.w
            << std::endl;

  // Writing final values to the file
  h5Helper.writeComFinal(comOnGPU.x, comOnGPU.y, comOnGPU.z, comOnGPU.w);
  h5Helper.writeParticleDataFinal();

  return 0;
}// end of main
//----------------------------------------------------------------------------------------------------------------------
