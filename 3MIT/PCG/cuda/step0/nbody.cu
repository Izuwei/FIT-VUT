#include "hip/hip_runtime.h"
/**
 * @File nbody.cu
 *
 * Implementation of the N-Body problem
 *
 * Paralelní programování na GPU (PCG 2021)
 * Projekt c. 1 (cuda)
 * Login: xsadil07
 */

#include <cmath>
#include <cfloat>
#include "nbody.h"

/**
 * CUDA kernel to calculate gravitation velocity
 * @param p       - particles
 * @param tmp_vel - temp array for velocities
 * @param N       - Number of particles
 * @param dt      - Size of the time step
 */
__global__ void calculate_gravitation_velocity(t_particles p, t_velocities tmp_vel, int N, float dt)
{
  unsigned int g_id = threadIdx.x + blockIdx.x * blockDim.x;

  if (g_id >= N) {
    return;
  }

  // Setup registers to avoid repeated access into global memory
  float F, r, dx, dy, dz;

  float vx = 0;
  float vy = 0;
  float vz = 0;

  float p1_w = p.weights[g_id];
  float p1_x = p.positions_x[g_id];
  float p1_y = p.positions_y[g_id];
  float p1_z = p.positions_z[g_id];

  // Loop over particles
  for (unsigned int p_id = 0; p_id < N; p_id++) {
    dx = p1_x - p.positions_x[p_id];
    dy = p1_y - p.positions_y[p_id];
    dz = p1_z - p.positions_z[p_id];

    r = sqrt(dx*dx + dy*dy + dz*dz);

    F = -G * p1_w * p.weights[p_id] / (r * r + FLT_MIN);

    vx += (r > COLLISION_DISTANCE) ? ((F * dx / (r + FLT_MIN)) * dt / p1_w) : 0.0f;
    vy += (r > COLLISION_DISTANCE) ? ((F * dy / (r + FLT_MIN)) * dt / p1_w) : 0.0f;
    vz += (r > COLLISION_DISTANCE) ? ((F * dz / (r + FLT_MIN)) * dt / p1_w) : 0.0f;
  }

  // Store velocity
  tmp_vel.x[g_id] += vx;
  tmp_vel.y[g_id] += vy;
  tmp_vel.z[g_id] += vz;
}// end of calculate_gravitation_velocity
//----------------------------------------------------------------------------------------------------------------------

/**
 * CUDA kernel to calculate collision velocity
 * @param p       - particles
 * @param tmp_vel - temp array for velocities
 * @param N       - Number of particles
 * @param dt      - Size of the time step
 */
__global__ void calculate_collision_velocity(t_particles p, t_velocities tmp_vel, int N, float dt)
{ 
  unsigned int g_id = threadIdx.x + blockIdx.x * blockDim.x;

  if (g_id >= N) {
    return;
  }

  // Setup registers to avoid repeated access into global memory
  float p2_w, r, dx, dy, dz;
    
  float vx = 0;
  float vy = 0;
  float vz = 0;

  float p1_w = p.weights[g_id];
  float p1_x = p.positions_x[g_id];
  float p1_y = p.positions_y[g_id];
  float p1_z = p.positions_z[g_id];
  float p1_vx = p.velocities_x[g_id];
  float p1_vy = p.velocities_y[g_id];
  float p1_vz = p.velocities_z[g_id];

  // Loop over particles
  for (unsigned int p_id = 0; p_id < N; p_id++) {
    p2_w = p.weights[p_id];

    dx = p1_x - p.positions_x[p_id];
    dy = p1_y - p.positions_y[p_id];
    dz = p1_z - p.positions_z[p_id];

    r = sqrt(dx*dx + dy*dy + dz*dz);
    
    vx += (r > 0.0f && r < COLLISION_DISTANCE) ? (((p1_w * p1_vx - p2_w * p1_vx + 2 * p2_w * p.velocities_x[p_id]) /
            (p1_w + p2_w)) - p1_vx) : 0.0f;
    vy += (r > 0.0f && r < COLLISION_DISTANCE) ? (((p1_w * p1_vy - p2_w * p1_vy + 2 * p2_w * p.velocities_y[p_id]) /
            (p1_w + p2_w)) - p1_vy) : 0.0f;
    vz += (r > 0.0f && r < COLLISION_DISTANCE) ? (((p1_w * p1_vz - p2_w * p1_vz + 2 * p2_w * p.velocities_z[p_id]) /
            (p1_w + p2_w)) - p1_vz) : 0.0f;
  }

  // Store velocity
  tmp_vel.x[g_id] += vx;
  tmp_vel.y[g_id] += vy;
  tmp_vel.z[g_id] += vz;
}// end of calculate_collision_velocity
//----------------------------------------------------------------------------------------------------------------------

/**
 * CUDA kernel to update particles
 * @param p       - particles
 * @param tmp_vel - temp array for velocities
 * @param N       - Number of particles
 * @param dt      - Size of the time step
 */
__global__ void update_particle(t_particles p, t_velocities tmp_vel, int N, float dt)
{
  unsigned int g_id = threadIdx.x + blockIdx.x * blockDim.x;

  if (g_id < N) {
    p.velocities_x[g_id] += tmp_vel.x[g_id];
    p.velocities_y[g_id] += tmp_vel.y[g_id];
    p.velocities_z[g_id] += tmp_vel.z[g_id];

    p.positions_x[g_id] += p.velocities_x[g_id] * dt;
    p.positions_y[g_id] += p.velocities_y[g_id] * dt;
    p.positions_z[g_id] += p.velocities_z[g_id] * dt;
  }
}// end of update_particle
//----------------------------------------------------------------------------------------------------------------------

/**
 * CUDA kernel to update particles
 * @param p       - particles
 * @param comX    - pointer to a center of mass position in X
 * @param comY    - pointer to a center of mass position in Y
 * @param comZ    - pointer to a center of mass position in Z
 * @param comW    - pointer to a center of mass weight
 * @param lock    - pointer to a user-implemented lock
 * @param N       - Number of particles
 */
__global__ void centerOfMass(t_particles p, float* comX, float* comY, float* comZ, float* comW, int* lock, const int N)
{

}// end of centerOfMass
//----------------------------------------------------------------------------------------------------------------------

/**
 * CPU implementation of the Center of Mass calculation
 * @param particles - All particles in the system
 * @param N         - Number of particles
 */
__host__ float4 centerOfMassCPU(MemDesc& memDesc)
{
  float4 com = {0 ,0, 0, 0};

  for(int i = 0; i < memDesc.getDataSize(); i++)
  {
    // Calculate the vector on the line connecting current body and most recent position of center-of-mass
    const float dx = memDesc.getPosX(i) - com.x;
    const float dy = memDesc.getPosY(i) - com.y;
    const float dz = memDesc.getPosZ(i) - com.z;

    // Calculate weight ratio only if at least one particle isn't massless
    const float dw = ((memDesc.getWeight(i) + com.w) > 0.0f)
                          ? ( memDesc.getWeight(i) / (memDesc.getWeight(i) + com.w)) : 0.0f;

    // Update position and weight of the center-of-mass according to the weight ration and vector
    com.x += dx * dw;
    com.y += dy * dw;
    com.z += dz * dw;
    com.w += memDesc.getWeight(i);
  }
  return com;
}// enf of centerOfMassCPU
//----------------------------------------------------------------------------------------------------------------------
